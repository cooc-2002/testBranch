#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "superResolution/SR_GPU2.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include "superResolution/SR_GPU_Functions.h"

//texture<unsigned char,2, hipReadModeNormalizedFloat> image_TM;
texture<unsigned char,2, hipReadModeElementType> image_TM;
texture<unsigned char,2, hipReadModeElementType> gray_TM;

__global__ void rgbToGray(unsigned char* input, int height, int width, float* output){	// input:RGB image, output:gray_image
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int index = x*width+y;

	if (x>=0 && x<height && y>= 0 && y<width)
		output[index] = float(16.0f + (65.738f*input[index*3]+129.057f*input[index*3 +1]+25.064f*input[index*3 + 2])/256.0f +0.5f);
}

__global__ void Hessian(float* input, int height, int width, float* output){ //input:gray_image, output: e_vec
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	float dxx_Hessian, dyy_Hessian, dxy_Hessian, e_val;

	if (x>=1 && x<height-1 && y>= 1 && y<width-1){
		dxx_Hessian = input[x*width + y+1] + input[x*width + y-1] - 2*input[x*width + y];
		dyy_Hessian = input[(x+1)*width + y] + input[(x-1)*width + y] - 2*input[x*width + y];
		dxy_Hessian = (input[(x-1)*width + (y-1)] + input[(x+1)*width + (y+1)] - input[(x+1)*width + (y-1)] - input[(x-1)*width + (y+1)])/4.0f;

		e_val = 0.5f * (dxx_Hessian + dyy_Hessian + sqrtf((dxx_Hessian - dyy_Hessian)*(dxx_Hessian - dyy_Hessian) + 4*dxy_Hessian*dxy_Hessian));
		if (abs(dxy_Hessian) <= 0.01 || abs(e_val - dxx_Hessian) <= 0.01){
			output[2*(x*width + y)] = 1;
			output[2*(x*width + y)+1] = 0;
		}else{
			output[2*(x*width + y)] = dxy_Hessian;
			output[2*(x*width + y)+1] = e_val - dxx_Hessian;
		}
	}
}

__global__ void Interpolation(unsigned char* input, float* e_vec, int height, int width, unsigned char *output){ // input:image, output: result_image
	__shared__ float shared[Thread_Size*3*2][3];

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	//////////////////////////////////////////////////
	//////// I. Save shared memory part ////////
	//////////////////////////////////////////////////

	shared[6*(threadIdx.y/3)+(threadIdx.y%3)][0] = input[3*(width*x+(blockIdx.y*blockDim.y))+threadIdx.y];
	shared[6*((threadIdx.y+Thread_Size)/3)+((threadIdx.y+Thread_Size)%3)][0] = input[3*(width*x+(blockIdx.y*blockDim.y))+(threadIdx.y+Thread_Size)];
	shared[6*((threadIdx.y+Thread_Size*2)/3)+((threadIdx.y+Thread_Size*2)%3)][0] = input[3*(width*x+(blockIdx.y*blockDim.y))+(threadIdx.y+Thread_Size*2)];

	shared[6*(threadIdx.y/3)+(threadIdx.y%3)][2] = input[3*(width*(x+1)+(blockIdx.y*blockDim.y))+threadIdx.y];
	shared[6*((threadIdx.y+Thread_Size)/3)+((threadIdx.y+Thread_Size)%3)][2] = input[3*(width*(x+1)+(blockIdx.y*blockDim.y))+(threadIdx.y+Thread_Size)];
	shared[6*((threadIdx.y+Thread_Size*2)/3)+((threadIdx.y+Thread_Size*2)%3)][2] = input[3*(width*(x+1)+(blockIdx.y*blockDim.y))+(threadIdx.y+Thread_Size*2)];

//	shared[6*(threadIdx.y/3)+(threadIdx.y%3)][2] = tex2D(image_TM,3*y+0,x);
//	shared[6*((threadIdx.y+Thread_Size)/3)+((threadIdx.y+Thread_Size)%3)][2] = tex2D(image_TM,3*y+1,x);
//	shared[6*((threadIdx.y+Thread_Size*2)/3)+((threadIdx.y+Thread_Size*2)%3)][2] = tex2D(image_TM,3*y+2,x);

	__syncthreads();

	///////////////////////////////////////////////////
	//////////// II. Processing part ////////////
	///////////////////////////////////////////////////

	float a=abs(e_vec[2*(x*width + y)]);
	float b=abs(e_vec[2*(x*width + y)+1]);
	float ori = b/a;

	float P1, P2;

	if (ori>=1){
		P1 = (b-a)/(2*b);
		P2 = (a+b)/(2*b);
	}
	else if(ori>=0 && ori< 1){
		P1 = (a+b)/(2*a);
		P2 = (a-b)/(2*a);
	}
	else if(ori>=-1 && ori< 0){
		P1 = (a+b)/(2*a);
		P2 = (a-b)/(2*a);
	}
	else{
		P1 = (a+b)/(2*b);
		P2 = (b-a)/(2*b);
	}

	if (abs(ori)>=1){
		if (threadIdx.y < Thread_Size-1){
			shared[threadIdx.y*6 + 3][1] = 0.5 * P1 * (shared[threadIdx.y*6 + 0][0] + shared[threadIdx.y*6 + 6 + 0][2]) + 0.5 * P2 * (shared[threadIdx.y*6 + 0][2] + shared[threadIdx.y*6 + 6 + 0][0]);
			shared[threadIdx.y*6 + 4][1] = 0.5 * P1 * (shared[threadIdx.y*6 + 1][0] + shared[threadIdx.y*6 + 6 + 1][2]) + 0.5 * P2 * (shared[threadIdx.y*6 + 1][2] + shared[threadIdx.y*6 + 6 + 1][0]);
			shared[threadIdx.y*6 + 5][1] = 0.5 * P1 * (shared[threadIdx.y*6 + 2][0] + shared[threadIdx.y*6 + 6 + 2][2]) + 0.5 * P2 * (shared[threadIdx.y*6 + 2][2] + shared[threadIdx.y*6 + 6 + 2][0]);
		}
		else{
			shared[threadIdx.y*6 + 3][1] = P1 * shared[threadIdx.y*6 + 0][0] + P2 * shared[threadIdx.y*6 + 0][2];
			shared[threadIdx.y*6 + 4][1] = P1 * shared[threadIdx.y*6 + 1][0] + P2 * shared[threadIdx.y*6 + 1][2];
			shared[threadIdx.y*6 + 5][1] = P1 * shared[threadIdx.y*6 + 2][0] + P2 * shared[threadIdx.y*6 + 2][2];
		}
	}
	else{
		if (threadIdx.y < Thread_Size-1){
			shared[threadIdx.y*6 + 3][1] = 0.5 * P1 * (shared[threadIdx.y*6 + 0][2] + shared[threadIdx.y*6 + 6 + 0][0]) + 0.5 * P2 * (shared[threadIdx.y*6 + 6 + 0][2] + shared[threadIdx.y*6 + 0][0]);
			shared[threadIdx.y*6 + 4][1] = 0.5 * P1 * (shared[threadIdx.y*6 + 1][2] + shared[threadIdx.y*6 + 6 + 1][0]) + 0.5 * P2 * (shared[threadIdx.y*6 + 6 + 1][2] + shared[threadIdx.y*6 + 1][0]);
			shared[threadIdx.y*6 + 5][1] = 0.5 * P1 * (shared[threadIdx.y*6 + 2][2] + shared[threadIdx.y*6 + 6 + 2][0]) + 0.5 * P2 * (shared[threadIdx.y*6 + 6 + 2][2] + shared[threadIdx.y*6 + 2][0]);
		}
		else{
			shared[threadIdx.y*6 + 3][1] = P1 * (shared[threadIdx.y*6 + 0][2]) + P2 * shared[threadIdx.y*6 + 0][0];
			shared[threadIdx.y*6 + 4][1] = P1 * (shared[threadIdx.y*6 + 1][2]) + P2 * shared[threadIdx.y*6 + 1][0];
			shared[threadIdx.y*6 + 5][1] = P1 * (shared[threadIdx.y*6 + 2][2]) + P2 * shared[threadIdx.y*6 + 2][0];
		}
	}

	__syncthreads();

	// horizontal

	if (threadIdx.y < Thread_Size-1){

		a = sqrtf((shared[threadIdx.y*6 + 0][0] - shared[threadIdx.y*6 + 3][1]) * (shared[threadIdx.y*6 + 0][0] - shared[threadIdx.y*6 + 3][1]) +
				   (shared[threadIdx.y*6 + 1][0] - shared[threadIdx.y*6 + 4][1]) * (shared[threadIdx.y*6 + 1][0] - shared[threadIdx.y*6 + 4][1]) +
				   (shared[threadIdx.y*6 + 2][0] - shared[threadIdx.y*6 + 5][1]) * (shared[threadIdx.y*6 + 2][0] - shared[threadIdx.y*6 + 5][1]) );
		b = sqrtf((shared[threadIdx.y*6 + 6 + 0][0] - shared[threadIdx.y*6 + 3][1]) * (shared[threadIdx.y*6 + 6 + 0][0] - shared[threadIdx.y*6 + 3][1]) +
				   (shared[threadIdx.y*6 + 6 + 1][0] - shared[threadIdx.y*6 + 4][1]) * (shared[threadIdx.y*6 + 6 + 1][0] - shared[threadIdx.y*6 + 4][1]) +
				   (shared[threadIdx.y*6 + 6 + 2][0] - shared[threadIdx.y*6 + 5][1]) * (shared[threadIdx.y*6 + 6 + 2][0] - shared[threadIdx.y*6 + 5][1]) );

		if (a==0 && b==0){
			P1 = 0.5;
			P2 = 0.5;
		}
		else if (a!=0 && b==0){
			P1 = 0;
			P2 = 1;
		}
		else if(a==0 && b!=0){
			P1 = 1;
			P2 = 0;
		}
		else{
			P1 = b/(a+b);
			P2 = a/(a+b);
		}

		shared[threadIdx.y*6 + 3][0] = P1 * shared[threadIdx.y*6 + 0][0] + P2 * shared[threadIdx.y*6 + 6 + 0][0];
		shared[threadIdx.y*6 + 4][0] = P1 * shared[threadIdx.y*6 + 1][0] + P2 * shared[threadIdx.y*6 + 6 + 1][0];
		shared[threadIdx.y*6 + 5][0] = P1 * shared[threadIdx.y*6 + 2][0] + P2 * shared[threadIdx.y*6 + 6 + 2][0];
	}
	else{
		shared[threadIdx.y*6 + 3][0] = shared[threadIdx.y*6 + 0][0];
		shared[threadIdx.y*6 + 4][0] = shared[threadIdx.y*6 + 1][0];
		shared[threadIdx.y*6 + 5][0] = shared[threadIdx.y*6 + 2][0];
	}

	// vertical

	if (threadIdx.y < Thread_Size-1){

		a = sqrtf((shared[threadIdx.y*6 + 0][2] - shared[threadIdx.y*6 + 3][1]) * (shared[threadIdx.y*6 + 0][2] - shared[threadIdx.y*6 + 3][1]) +
				   (shared[threadIdx.y*6 + 1][2] - shared[threadIdx.y*6 + 4][1]) * (shared[threadIdx.y*6 + 1][2] - shared[threadIdx.y*6 + 4][1]) +
				   (shared[threadIdx.y*6 + 2][2] - shared[threadIdx.y*6 + 5][1]) * (shared[threadIdx.y*6 + 2][2] - shared[threadIdx.y*6 + 5][1]) );
		b = sqrtf((shared[threadIdx.y*6 + 0][0] - shared[threadIdx.y*6 + 3][1]) * (shared[threadIdx.y*6 + 0][0] - shared[threadIdx.y*6 + 3][1]) +
				   (shared[threadIdx.y*6 + 1][0] - shared[threadIdx.y*6 + 4][1]) * (shared[threadIdx.y*6 + 1][0] - shared[threadIdx.y*6 + 4][1]) +
				   (shared[threadIdx.y*6 + 2][0] - shared[threadIdx.y*6 + 5][1]) * (shared[threadIdx.y*6 + 2][0] - shared[threadIdx.y*6 + 5][1]) );

		if (a==0 && b==0){
			P1 = 0.5;
			P2 = 0.5;
		}
		else if (a!=0 && b==0){
			P1 = 1;
			P2 = 0;
		}
		else if(a==0 && b!=0){
			P1 = 0;
			P2 = 1;
		}
		else{
			P1 = b/(a+b);
			P2 = a/(a+b);
		}

		shared[threadIdx.y*6 + 0][1] = P1 * shared[threadIdx.y*6 + 0][2] + P2 * shared[threadIdx.y*6 + 0][0];
		shared[threadIdx.y*6 + 1][1] = P1 * shared[threadIdx.y*6 + 1][2] + P2 * shared[threadIdx.y*6 + 1][0];
		shared[threadIdx.y*6 + 2][1] = P1 * shared[threadIdx.y*6 + 2][2] + P2 * shared[threadIdx.y*6 + 2][0];
	}
	else{
		shared[threadIdx.y*6 + 0][1] = shared[threadIdx.y*6 + 0][0];
		shared[threadIdx.y*6 + 1][1] = shared[threadIdx.y*6 + 1][0];
		shared[threadIdx.y*6 + 2][1] = shared[threadIdx.y*6 + 2][0];
	}
	__syncthreads();

	//////////////////////////////////////////////////
	/////// III. Load shared memory part ///////
	//////////////////////////////////////////////////

	if(y < width){
		output[3*(2*width*2*x+2*(blockIdx.y*blockDim.y))+(threadIdx.y)] = shared[threadIdx.y][0];
		output[3*(2*width*2*x+2*(blockIdx.y*blockDim.y))+(threadIdx.y+Thread_Size)] = shared[threadIdx.y+Thread_Size][0];
		output[3*(2*width*2*x+2*(blockIdx.y*blockDim.y))+(threadIdx.y+Thread_Size*2)] = shared[threadIdx.y+Thread_Size*2][0];

		output[3*(2*width*2*x+2*(blockIdx.y*blockDim.y))+(threadIdx.y+Thread_Size*3)] = shared[threadIdx.y+Thread_Size*3][0];
		output[3*(2*width*2*x+2*(blockIdx.y*blockDim.y))+(threadIdx.y+Thread_Size*4)] = shared[threadIdx.y+Thread_Size*4][0];
		output[3*(2*width*2*x+2*(blockIdx.y*blockDim.y))+(threadIdx.y+Thread_Size*5)] = shared[threadIdx.y+Thread_Size*5][0];

		output[3*(2*width*(2*x+1)+2*(blockIdx.y*blockDim.y))+(threadIdx.y)] = shared[threadIdx.y][1];
		output[3*(2*width*(2*x+1)+2*(blockIdx.y*blockDim.y))+(threadIdx.y+Thread_Size)] = shared[threadIdx.y+Thread_Size][1];
		output[3*(2*width*(2*x+1)+2*(blockIdx.y*blockDim.y))+(threadIdx.y+Thread_Size*2)] = shared[threadIdx.y+Thread_Size*2][1];

		output[3*(2*width*(2*x+1)+2*(blockIdx.y*blockDim.y))+(threadIdx.y+Thread_Size*3)] = shared[threadIdx.y+Thread_Size*3][1];
		output[3*(2*width*(2*x+1)+2*(blockIdx.y*blockDim.y))+(threadIdx.y+Thread_Size*4)] = shared[threadIdx.y+Thread_Size*4][1];
		output[3*(2*width*(2*x+1)+2*(blockIdx.y*blockDim.y))+(threadIdx.y+Thread_Size*5)] = shared[threadIdx.y+Thread_Size*5][1];
	}
}

__global__ void ucharCopy(float* d_result_img, int height, int width, unsigned char* d_result_img_uchar){
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x<=(2*height-1) && y<=(2*width-1)){
		d_result_img_uchar[3*(x*(2*width)+y)  ] = d_result_img[3*(x*(2*width)+y)  ];
		d_result_img_uchar[3*(x*(2*width)+y)+1] = d_result_img[3*(x*(2*width)+y)+1];
		d_result_img_uchar[3*(x*(2*width)+y)+2] = d_result_img[3*(x*(2*width)+y)+2];
	}
}








//////////////////////////////////////
/////Texture Memory Function/////
//////////////////////////////////////

__global__ void rgbToGray_tex(int height, int width, float* output){	// input:RGB image, output:gray_image
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int index = x*width+y;

	if (x>=0 && x<height && y>= 0 && y<width)
		//tex2D(gray_TM, y, x) = float(16.0f + (65.738f*tex2D(image_TM, 3*y+0, x)+129.057f*tex2D(image_TM, 3*y+1, x)+25.064f*tex2D(image_TM, 3*y+2, x))/256.0f +0.5f);
		output[index] = float(16.0f + (65.738f*tex2D(image_TM, 3*y+0, x)+129.057f*tex2D(image_TM, 3*y+1, x)+25.064f*tex2D(image_TM, 3*y+2, x))/256.0f +0.5f);
}

__global__ void Hessian_tex(int height, int width, float* output){ //input:gray_image, output: e_vec
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	float dxx_Hessian, dyy_Hessian, dxy_Hessian, e_val;

	if (x>=1 && x<height-1 && y>= 1 && y<width-1){
		dxx_Hessian = tex2D(gray_TM, y+1, x) + tex2D(gray_TM, y-1, x) - 2*tex2D(gray_TM, y, x);
		dyy_Hessian = tex2D(gray_TM, y, x+1) + tex2D(gray_TM, y, x-1) - 2*tex2D(gray_TM, y, x);
		dxy_Hessian = (tex2D(gray_TM, y-1, x-1) + tex2D(gray_TM, y+1, x+1) - tex2D(gray_TM, y-1, x+1) - tex2D(gray_TM, y+1, x-1))/4.0f;

		e_val = 0.5f * (dxx_Hessian + dyy_Hessian + sqrtf((dxx_Hessian - dyy_Hessian)*(dxx_Hessian - dyy_Hessian) + 4*dxy_Hessian*dxy_Hessian));
		if (abs(dxy_Hessian) <= 0.01 || abs(e_val - dxx_Hessian) <= 0.01){
			output[2*(x*width + y)] = 1;
			output[2*(x*width + y)+1] = 0;
		}else{
			output[2*(x*width + y)] = dxy_Hessian;
			output[2*(x*width + y)+1] = e_val - dxx_Hessian;
		}
	}
}

__global__ void Interpolation_tex(float* e_vec, int height, int width, unsigned char *output){ // input:image, output: result_image
	__shared__ float shared[Thread_Size*3*2];

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	//////////////////////////////////////////////////
	//////// I. Save shared memory part ////////
	//////////////////////////////////////////////////

	shared[6*(threadIdx.y/3)+(threadIdx.y%3)] = tex2D(image_TM, 3*(blockIdx.y*blockDim.y)+threadIdx.y, x);
	shared[6*((threadIdx.y+Thread_Size)/3)+((threadIdx.y+Thread_Size)%3)] = tex2D(image_TM, 3*(blockIdx.y*blockDim.y)+threadIdx.y+Thread_Size, x);
	shared[6*((threadIdx.y+Thread_Size*2)/3)+((threadIdx.y+Thread_Size*2)%3)] = tex2D(image_TM, 3*(blockIdx.y*blockDim.y)+threadIdx.y+Thread_Size*2, x);

	__syncthreads();

	///////////////////////////////////////////////////
	//////////// II. Processing part ////////////
	///////////////////////////////////////////////////

	float a=abs(e_vec[2*(x*width + y)]);
	float b=abs(e_vec[2*(x*width + y)+1]);
	float ori = b/a;

	float P1, P2;

	if (ori>=1){
		P2 = (b-a)/(2*b);
		P1 = (a+b)/(2*b);
	}
	else if(ori>=0 && ori< 1){
		P1 = (a+b)/(2*a);
		P2 = (a-b)/(2*a);
	}
	else if(ori>=-1 && ori< 0){
		P1 = (a+b)/(2*a);
		P2 = (a-b)/(2*a);
	}
	else{
		P2 = (a+b)/(2*b);
		P1 = (b-a)/(2*b);
	}

	float diagonal[3];
	diagonal[0] = 0.5 * P1 * (tex2D(image_TM, 3*y+0, x) + tex2D(image_TM, 3*(y+1)+0, x+1)) + 0.5 * P2 * (tex2D(image_TM, 3*y+0, x+1) + tex2D(image_TM, 3*(y+1)+0, x));
	diagonal[1] = 0.5 * P1 * (tex2D(image_TM, 3*y+1, x) + tex2D(image_TM, 3*(y+1)+1, x+1)) + 0.5 * P2 * (tex2D(image_TM, 3*y+1, x+1) + tex2D(image_TM, 3*(y+1)+1, x));
	diagonal[2] = 0.5 * P1 * (tex2D(image_TM, 3*y+2, x) + tex2D(image_TM, 3*(y+1)+2, x+1)) + 0.5 * P2 * (tex2D(image_TM, 3*y+2, x+1) + tex2D(image_TM, 3*(y+1)+2, x));

	__syncthreads();

	// horizontal

	a = sqrtf((tex2D(image_TM, 3*y+0, x) - diagonal[0]) * (tex2D(image_TM, 3*y+0, x) - diagonal[0]) +
			   (tex2D(image_TM, 3*y+1, x) - diagonal[1]) * (tex2D(image_TM, 3*y+1, x) - diagonal[1]) +
			   (tex2D(image_TM, 3*y+2, x) - diagonal[2]) * (tex2D(image_TM, 3*y+2, x) - diagonal[2]));
	b = sqrtf((tex2D(image_TM, 3*(y+1)+0, x) - diagonal[0]) * (tex2D(image_TM, 3*(y+1)+0, x) - diagonal[0]) +
			   (tex2D(image_TM, 3*(y+1)+1, x) - diagonal[1]) * (tex2D(image_TM, 3*(y+1)+1, x) - diagonal[1]) +
			   (tex2D(image_TM, 3*(y+1)+2, x) - diagonal[2]) * (tex2D(image_TM, 3*(y+1)+2, x) - diagonal[2]));

	if (a==0 && b==0){
		P1 = 0.5;
		P2 = 0.5;
	}
	else if (a!=0 && b==0){
		P1 = 0;
		P2 = 1;
	}
	else if(a==0 && b!=0){
		P1 = 1;
		P2 = 0;
	}
	else{
		P1 = b/(a+b);
		P2 = a/(a+b);
	}

	shared[threadIdx.y*6 + 3] = P1 * tex2D(image_TM, 3*y+0, x) + P2 * tex2D(image_TM, 3*(y+1)+0, x);
	shared[threadIdx.y*6 + 4] = P1 * tex2D(image_TM, 3*y+1, x) + P2 * tex2D(image_TM, 3*(y+1)+1, x);
	shared[threadIdx.y*6 + 5] = P1 * tex2D(image_TM, 3*y+2, x) + P2 * tex2D(image_TM, 3*(y+1)+2, x);

	__syncthreads();

	if(y < width){
		output[3*(2*width*2*x+2*(blockIdx.y*blockDim.y))+(threadIdx.y)] = shared[threadIdx.y];
		output[3*(2*width*2*x+2*(blockIdx.y*blockDim.y))+(threadIdx.y+Thread_Size)] = shared[threadIdx.y+Thread_Size];
		output[3*(2*width*2*x+2*(blockIdx.y*blockDim.y))+(threadIdx.y+Thread_Size*2)] = shared[threadIdx.y+Thread_Size*2];

		output[3*(2*width*2*x+2*(blockIdx.y*blockDim.y))+(threadIdx.y+Thread_Size*3)] = shared[threadIdx.y+Thread_Size*3];
		output[3*(2*width*2*x+2*(blockIdx.y*blockDim.y))+(threadIdx.y+Thread_Size*4)] = shared[threadIdx.y+Thread_Size*4];
		output[3*(2*width*2*x+2*(blockIdx.y*blockDim.y))+(threadIdx.y+Thread_Size*5)] = shared[threadIdx.y+Thread_Size*5];
	}

//	if(threadIdx.y == 0 && blockIdx.y == 0 && threadIdx.x == 0 && blockIdx.x == 0 ){
//		printf("shared[0] = %f\n", shared[0]);
//		printf("shared[1] = %f\n", shared[1]);
//		printf("shared[2] = %f\n", shared[2]);
//		printf("shared[3] = %f\n", shared[3]);
//		printf("shared[4] = %f\n", shared[4]);
//		printf("shared[5] = %f\n", shared[5]);
//	}

	// vertical

	a = sqrtf((tex2D(image_TM, 3*y+0, x+1) - diagonal[0]) * (tex2D(image_TM, 3*y+0, x+1) - diagonal[0]) +
			   (tex2D(image_TM, 3*y+1, x+1) - diagonal[1]) * (tex2D(image_TM, 3*y+1, x+1) - diagonal[1]) +
			   (tex2D(image_TM, 3*y+2, x+1) - diagonal[2]) * (tex2D(image_TM, 3*y+2, x+1) - diagonal[2]) );
	b = sqrtf((tex2D(image_TM, 3*y+0, x) - diagonal[0]) * (tex2D(image_TM, 3*y+0, x) - diagonal[0]) +
			   (tex2D(image_TM, 3*y+1, x) - diagonal[1]) * (tex2D(image_TM, 3*y+1, x) - diagonal[1]) +
			   (tex2D(image_TM, 3*y+2, x) - diagonal[2]) * (tex2D(image_TM, 3*y+2, x) - diagonal[2]) );

	if (a==0 && b==0){
		P1 = 0.5;
		P2 = 0.5;
	}
	else if (a!=0 && b==0){
		P1 = 1;
		P2 = 0;
	}
	else if(a==0 && b!=0){
		P1 = 0;
		P2 = 1;
	}
	else{
		P1 = b/(a+b);
		P2 = a/(a+b);
	}

	__syncthreads();

	shared[threadIdx.y*6 + 0] = P1 * tex2D(image_TM, 3*y+0, x+1) + P2 * tex2D(image_TM, 3*y+0, x);
	shared[threadIdx.y*6 + 1] = P1 * tex2D(image_TM, 3*y+1, x+1) + P2 * tex2D(image_TM, 3*y+1, x);
	shared[threadIdx.y*6 + 2] = P1 * tex2D(image_TM, 3*y+2, x+1) + P2 * tex2D(image_TM, 3*y+2, x);

	shared[threadIdx.y*6 + 3] = diagonal[0];
	shared[threadIdx.y*6 + 4] = diagonal[1];
	shared[threadIdx.y*6 + 5] = diagonal[2];

	__syncthreads();


	//////////////////////////////////////////////////
	/////// III. Load shared memory part ///////
	//////////////////////////////////////////////////

	if(y < width){
		output[3 * (2 * width*(2 * x + 1) + 2 * (blockIdx.y*blockDim.y)) + (threadIdx.y)] = shared[threadIdx.y];
		output[3*(2*width*(2*x+1)+2*(blockIdx.y*blockDim.y))+(threadIdx.y+Thread_Size)] = shared[threadIdx.y+Thread_Size];
		output[3*(2*width*(2*x+1)+2*(blockIdx.y*blockDim.y))+(threadIdx.y+Thread_Size*2)] = shared[threadIdx.y+Thread_Size*2];

		output[3*(2*width*(2*x+1)+2*(blockIdx.y*blockDim.y))+(threadIdx.y+Thread_Size*3)] = shared[threadIdx.y+Thread_Size*3];
		output[3*(2*width*(2*x+1)+2*(blockIdx.y*blockDim.y))+(threadIdx.y+Thread_Size*4)] = shared[threadIdx.y+Thread_Size*4];
		output[3*(2*width*(2*x+1)+2*(blockIdx.y*blockDim.y))+(threadIdx.y+Thread_Size*5)] = shared[threadIdx.y+Thread_Size*5];
	}
}


void bindTexture(unsigned char* image, int width, int height){
	// set up the CUDA array
	hipChannelFormatDesc desc = hipCreateChannelDesc<unsigned char>();
	hipArray* texArray = 0;
	hipMallocArray(&texArray, &desc, 3*width, height);
	hipMemcpyToArray(texArray, 0,0, image, 3*width*height*sizeof(unsigned char),hipMemcpyHostToDevice);

	// specify mutable texture reference parameters
//	image_TM.normalized = true;
//	image_TM.filterMode = hipFilterModeLinear;
//	image_TM.addressMode[0] = hipAddressModeBorder;
//	image_TM.addressMode[1] = hipAddressModeBorder;

	// bind texture reference to array
	hipBindTextureToArray(image_TM, texArray);
}

void bindTexture_gray(float* d_gray_img, int width, int height){
	// set up the CUDA array
	hipChannelFormatDesc desc = hipCreateChannelDesc<unsigned char>();
	hipArray* texArray = 0;
	hipMallocArray(&texArray, &desc, width, height);
	hipMemcpyToArray(texArray, 0,0, d_gray_img, width*height*sizeof(unsigned char),hipMemcpyHostToDevice);

	// specify mutable texture reference parameters
//	image_TM.normalized = true;
//	image_TM.filterMode = hipFilterModeLinear;
//	image_TM.addressMode[0] = hipAddressModeBorder;
//	image_TM.addressMode[1] = hipAddressModeBorder;

	// bind texture reference to array
	hipBindTextureToArray(gray_TM, texArray);
}
