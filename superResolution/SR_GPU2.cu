#include <stdio.h>
#include <stdlib.h>
#include "superResolution/SR_GPU2.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include "superResolution/SR_GPU_Functions.h"

SR_GPU2::SR_GPU2() {
	d_gray_img = NULL;
	d_e_vec = NULL;
	d_result_img = NULL;

	hipSetDevice(0);
	hipProfilerStart();
	hipDeviceSynchronize();
}

SR_GPU2::~SR_GPU2() {
	if (d_gray_img != NULL) hipFree(d_gray_img);
	if (d_e_vec != NULL) hipFree(d_e_vec);
	if (d_result_img != NULL) hipFree(d_result_img);
}

void SR_GPU2::setImage(unsigned char* _image, int _width, int _height) {
	image = _image;
	width = _width;
	height = _height;

	if (pixelNum != width*height) {
		pixelNum = width*height;

		if (d_gray_img != NULL) hipFree(d_gray_img);
		if (d_e_vec != NULL) hipFree(d_e_vec);
		if (d_result_img != NULL) hipFree(d_result_img);
		if (output != NULL) free(output);

		hipMalloc((void**)&d_gray_img, height*width*sizeof(float));
		hipMalloc((void**)&d_e_vec, height*width * 2 * sizeof(float));
		hipMalloc((void**)&d_result_img, (2 * height + 1) * 2 * width * 3 * sizeof(unsigned char));
		output = (unsigned char*)malloc(2 * height * 2 * width * 3 * sizeof(unsigned char));
	}
}

unsigned char* SR_GPU2::perform() {
	// bind texture reference to array
	bindTexture(image, width, height);

	dim3 threads, numBlocks;
	threads.x = 1;
	threads.y = Thread_Size;
	numBlocks.x = height;
	numBlocks.y = width / threads.y + 1;

	rgbToGray_tex << <numBlocks, threads >> >(height, width, d_gray_img);
	hipDeviceSynchronize();

	Hessian << <numBlocks, threads >> >(d_gray_img, height, width, d_e_vec);
	hipDeviceSynchronize();

	Interpolation_tex << <numBlocks, threads >> >(d_e_vec, height, width, d_result_img);
	hipDeviceSynchronize();
	hipMemcpy(output, d_result_img, 2 * height * 2 * width * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost); //result_img

	hipProfilerStop();

	return output;
}

