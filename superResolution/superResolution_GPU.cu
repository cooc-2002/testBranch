#include "hip/hip_runtime.h"
#include <stdio.h>
#include "superResolution/superResolution_GPU.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <chrono>


#define Memory_Size 1024
#define Block_Size 8

__global__ void rgbToGray(unsigned char* input, int height, int width, float* output);
__global__ void Hessian(float* input, int height, int width, float* output);
__global__ void Interpolation(unsigned char* input, float* e_vec, int height, int width, unsigned char* output);
__global__ void ucharCopy(float* d_result_img, int height, int width,unsigned char* d_result_img_uchar);

unsigned char* superResolution_GPU(unsigned char* image0, int height, int width, int file_number){
	float sumTime;

	// Number of CUDA devices
	int devCount;
	hipGetDeviceCount(&devCount);

	// Setting
	for (int i=0; i<devCount; ++i){
		hipSetDevice(i);
		hipDeviceReset();
	}
	hipDeviceSynchronize();

	hipProfilerStart();
	hipDeviceSynchronize();

	std::chrono::system_clock::time_point startTime;
	std::chrono::microseconds endTime;
	startTime = std::chrono::system_clock::now();

	hipStream_t stream1;
	hipStreamCreate( &stream1);

	dim3 threads, numBlocks;
	threads.x = 1;
	threads.y = 128;
	numBlocks.x = height;
	numBlocks.y = width/threads.y + 1;

	///////////////////////////////////////
	//////    Convert RGB to gray    //////
	///////////////////////////////////////

	unsigned char* d_img;
	float* d_gray_img;
	hipMalloc((void**) &d_img, (height+2)*width*3*sizeof(unsigned char));
	hipMalloc((void**) &d_gray_img, height*width*sizeof(float));
	hipMemcpy(d_img, image0,  height*width*3*sizeof(unsigned char), hipMemcpyHostToDevice);
	rgbToGray<<<numBlocks,threads>>>(d_img, height, width, d_gray_img);
	hipDeviceSynchronize();

	/////////////////////////////////////
	////     Hessian Analysis      //////
	/////////////////////////////////////

	float* d_e_vec;
	hipMalloc((void**) &d_e_vec, height*width*2*sizeof(float));
	Hessian<<<numBlocks,threads>>>(d_gray_img, height, width, d_e_vec);
	hipDeviceSynchronize();

	/////////////////////////////////////
	////       Interpolation       //////
	/////////////////////////////////////

	unsigned char * d_result_img;
	hipMalloc((void**) &d_result_img, (2*height+1)*2*width*3*sizeof(unsigned char));
	Interpolation<<<numBlocks,threads>>>(d_img, d_e_vec, height, width, d_result_img);
	unsigned char* output = (unsigned char*)malloc(2*height*2*width*3*sizeof(unsigned char));
	hipDeviceSynchronize();
	hipMemcpy(output, d_result_img,  2*height*2*width*3*sizeof(unsigned char), hipMemcpyDeviceToHost); //result_img

	hipFree(d_img);
	hipFree(d_gray_img);
	hipFree(d_e_vec);
	hipFree(d_result_img);

	endTime = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now() - startTime);
	sumTime += endTime.count()/1000.0f;
	printf("elapsed time = %3.3f\n", endTime.count()/1000.0f);
	hipProfilerStop();

	return output;
}

//__global__ void rgbToGray(unsigned char* input, int height, int width, float* output){	// input:RGB image, output:gray_image
//	int x = threadIdx.x + blockIdx.x * blockDim.x;
//	int y = threadIdx.y + blockIdx.y * blockDim.y;
//	int index = x*width+y;
//
//	if (x>=0 && x<height && y>= 0 && y<width)
//		output[index] = float(16.0f + (65.738f*input[index*3]+129.057f*input[index*3 +1]+25.064f*input[index*3 + 2])/256.0f +0.5f);
//}
//
//__global__ void Hessian(float* input, int height, int width, float* output){ //input:gray_image, output: e_vec
//	int x = threadIdx.x + blockIdx.x * blockDim.x;
//	int y = threadIdx.y + blockIdx.y * blockDim.y;
//
//	float dxx_Hessian, dyy_Hessian, dxy_Hessian, e_val;
//
//	if (x>=1 && x<height-1 && y>= 1 && y<width-1){
//		dxx_Hessian = input[x*width + y+1] + input[x*width + y-1] - 2*input[x*width + y];
//		dyy_Hessian = input[(x+1)*width + y] + input[(x-1)*width + y] - 2*input[x*width + y];
//		dxy_Hessian = (input[(x-1)*width + (y-1)] + input[(x+1)*width + (y+1)] - input[(x+1)*width + (y-1)] - input[(x-1)*width + (y+1)])/4.0f;
//
//		e_val = 0.5f * (dxx_Hessian + dyy_Hessian + sqrtf((dxx_Hessian - dyy_Hessian)*(dxx_Hessian - dyy_Hessian) + 4*dxy_Hessian*dxy_Hessian));
//		if (abs(dxy_Hessian) <= 0.01 || abs(e_val - dxx_Hessian) <= 0.01){
//			output[2*(x*width + y)] = 1;
//			output[2*(x*width + y)+1] = 0;
//		}else{
//			output[2*(x*width + y)] = dxy_Hessian;
//			output[2*(x*width + y)+1] = e_val - dxx_Hessian;
//		}
//	}
//}
//
//__global__ void Interpolation(unsigned char* input, float* e_vec, int height, int width, unsigned char *output){ // input:image, output: result_image
//	__shared__ float shared[128*3*2][3];
//
//	int x = threadIdx.x + blockIdx.x * blockDim.x;
//	int y = threadIdx.y + blockIdx.y * blockDim.y;
//
//	//////////////////////////////////////////////////
//	//////// I. Save shared memory part ////////
//	//////////////////////////////////////////////////
//
//	shared[6*(threadIdx.y/3)+(threadIdx.y%3)][0] = input[3*(width*x+(blockIdx.y*blockDim.y))+threadIdx.y];
//	shared[6*((threadIdx.y+128)/3)+((threadIdx.y+128)%3)][0] = input[3*(width*x+(blockIdx.y*blockDim.y))+(threadIdx.y+128)];
//	shared[6*((threadIdx.y+256)/3)+((threadIdx.y+256)%3)][0] = input[3*(width*x+(blockIdx.y*blockDim.y))+(threadIdx.y+256)];
//
//	shared[6*(threadIdx.y/3)+(threadIdx.y%3)][2] = input[3*(width*(x+1)+(blockIdx.y*blockDim.y))+threadIdx.y];
//	shared[6*((threadIdx.y+128)/3)+((threadIdx.y+128)%3)][2] = input[3*(width*(x+1)+(blockIdx.y*blockDim.y))+(threadIdx.y+128)];
//	shared[6*((threadIdx.y+256)/3)+((threadIdx.y+256)%3)][2] = input[3*(width*(x+1)+(blockIdx.y*blockDim.y))+(threadIdx.y+256)];
//
//	__syncthreads();
//
//	///////////////////////////////////////////////////
//	//////////// II. Processing part ////////////
//	///////////////////////////////////////////////////
//
//	float a=abs(e_vec[2*(x*width + y)]);
//	float b=abs(e_vec[2*(x*width + y)+1]);
//	float ori = b/a;
//
//	float P1, P2;
//
//	if (ori>=1){
//		P1 = (b-a)/(2*b);
//		P2 = (a+b)/(2*b);
//	}
//	else if(ori>=0 && ori< 1){
//		P1 = (a+b)/(2*a);
//		P2 = (a-b)/(2*a);
//	}
//	else if(ori>=-1 && ori< 0){
//		P1 = (a+b)/(2*a);
//		P2 = (a-b)/(2*a);
//	}
//	else{
//		P1 = (a+b)/(2*b);
//		P2 = (b-a)/(2*b);
//	}
//
//	if (abs(ori)>=1){
//		// New algorithm
//		if (threadIdx.y < 127){
//			shared[threadIdx.y*6 + 3][1] = 0.5 * P1 * (shared[threadIdx.y*6 + 0][0] + shared[threadIdx.y*6 + 6 + 0][2]) + 0.5 * P2 * (shared[threadIdx.y*6 + 0][2] + shared[threadIdx.y*6 + 6 + 0][0]);
//			shared[threadIdx.y*6 + 4][1] = 0.5 * P1 * (shared[threadIdx.y*6 + 1][0] + shared[threadIdx.y*6 + 6 + 1][2]) + 0.5 * P2 * (shared[threadIdx.y*6 + 1][2] + shared[threadIdx.y*6 + 6 + 1][0]);
//			shared[threadIdx.y*6 + 5][1] = 0.5 * P1 * (shared[threadIdx.y*6 + 2][0] + shared[threadIdx.y*6 + 6 + 2][2]) + 0.5 * P2 * (shared[threadIdx.y*6 + 2][2] + shared[threadIdx.y*6 + 6 + 2][0]);
//		}
//		else{
//			shared[threadIdx.y*6 + 3][1] = P1 * shared[threadIdx.y*6 + 0][0] + P2 * shared[threadIdx.y*6 + 0][2];
//			shared[threadIdx.y*6 + 4][1] = P1 * shared[threadIdx.y*6 + 1][0] + P2 * shared[threadIdx.y*6 + 1][2];
//			shared[threadIdx.y*6 + 5][1] = P1 * shared[threadIdx.y*6 + 2][0] + P2 * shared[threadIdx.y*6 + 2][2];
//		}
//	}
//	else{
//		// New algorithm
//		if (threadIdx.y < 127){
//			shared[threadIdx.y*6 + 3][1] = 0.5 * P1 * (shared[threadIdx.y*6 + 0][2] + shared[threadIdx.y*6 + 6 + 0][0]) + 0.5 * P2 * (shared[threadIdx.y*6 + 6 + 0][2] + shared[threadIdx.y*6 + 0][0]);
//			shared[threadIdx.y*6 + 4][1] = 0.5 * P1 * (shared[threadIdx.y*6 + 1][2] + shared[threadIdx.y*6 + 6 + 1][0]) + 0.5 * P2 * (shared[threadIdx.y*6 + 6 + 1][2] + shared[threadIdx.y*6 + 1][0]);
//			shared[threadIdx.y*6 + 5][1] = 0.5 * P1 * (shared[threadIdx.y*6 + 2][2] + shared[threadIdx.y*6 + 6 + 2][0]) + 0.5 * P2 * (shared[threadIdx.y*6 + 6 + 2][2] + shared[threadIdx.y*6 + 2][0]);
//		}
//		else{
//			shared[threadIdx.y*6 + 3][1] = P1 * (shared[threadIdx.y*6 + 0][2]) + P2 * shared[threadIdx.y*6 + 0][0];
//			shared[threadIdx.y*6 + 4][1] = P1 * (shared[threadIdx.y*6 + 1][2]) + P2 * shared[threadIdx.y*6 + 1][0];
//			shared[threadIdx.y*6 + 5][1] = P1 * (shared[threadIdx.y*6 + 2][2]) + P2 * shared[threadIdx.y*6 + 2][0];
//		}
//	}
//
//	__syncthreads();
//
//	// horizontal
//
//	if (threadIdx.y < 127){
////		a = sqrtf(powf(shared[threadIdx.y*6 + 0][0] - shared[threadIdx.y*6 + 3][1],2) +
////				   powf(shared[threadIdx.y*6 + 1][0] - shared[threadIdx.y*6 + 4][1],2) +
////				   powf(shared[threadIdx.y*6 + 2][0] - shared[threadIdx.y*6 + 5][1],2));
////		b = sqrtf(powf(shared[threadIdx.y*6 + 6 + 0][0] - shared[threadIdx.y*6 + 3][1],2) +
////				   powf(shared[threadIdx.y*6 + 6 + 1][0] - shared[threadIdx.y*6 + 4][1],2) +
////				   powf(shared[threadIdx.y*6 + 6 + 2][0] - shared[threadIdx.y*6 + 5][1],2));
//
//		a = sqrtf((shared[threadIdx.y*6 + 0][0] - shared[threadIdx.y*6 + 3][1]) * (shared[threadIdx.y*6 + 0][0] - shared[threadIdx.y*6 + 3][1]) +
//				   (shared[threadIdx.y*6 + 1][0] - shared[threadIdx.y*6 + 4][1]) * (shared[threadIdx.y*6 + 1][0] - shared[threadIdx.y*6 + 4][1]) +
//				   (shared[threadIdx.y*6 + 2][0] - shared[threadIdx.y*6 + 5][1]) * (shared[threadIdx.y*6 + 2][0] - shared[threadIdx.y*6 + 5][1]) );
//		b = sqrtf((shared[threadIdx.y*6 + 6 + 0][0] - shared[threadIdx.y*6 + 3][1]) * (shared[threadIdx.y*6 + 6 + 0][0] - shared[threadIdx.y*6 + 3][1]) +
//				   (shared[threadIdx.y*6 + 6 + 1][0] - shared[threadIdx.y*6 + 4][1]) * (shared[threadIdx.y*6 + 6 + 1][0] - shared[threadIdx.y*6 + 4][1]) +
//				   (shared[threadIdx.y*6 + 6 + 2][0] - shared[threadIdx.y*6 + 5][1]) * (shared[threadIdx.y*6 + 6 + 2][0] - shared[threadIdx.y*6 + 5][1]) );
//
//		if (a==0 && b==0){
//			P1 = 0.5;
//			P2 = 0.5;
//		}
//		else if (a!=0 && b==0){
//			P1 = 0;
//			P2 = 1;
//		}
//		else if(a==0 && b!=0){
//			P1 = 1;
//			P2 = 0;
//		}
//		else{
//			P1 = b/(a+b);
//			P2 = a/(a+b);
//		}
//
//		shared[threadIdx.y*6 + 3][0] = P1 * shared[threadIdx.y*6 + 0][0] + P2 * shared[threadIdx.y*6 + 6 + 0][0];
//		shared[threadIdx.y*6 + 4][0] = P1 * shared[threadIdx.y*6 + 1][0] + P2 * shared[threadIdx.y*6 + 6 + 1][0];
//		shared[threadIdx.y*6 + 5][0] = P1 * shared[threadIdx.y*6 + 2][0] + P2 * shared[threadIdx.y*6 + 6 + 2][0];
//	}
//	else{
//		shared[threadIdx.y*6 + 3][0] = shared[threadIdx.y*6 + 0][0];
//		shared[threadIdx.y*6 + 4][0] = shared[threadIdx.y*6 + 1][0];
//		shared[threadIdx.y*6 + 5][0] = shared[threadIdx.y*6 + 2][0];
//	}
//
//	//__syncthreads();
//
//	// vertical
//
//	if (threadIdx.y < 127){
////		a = sqrtf(powf(shared[threadIdx.y*6 + 0][2] - shared[threadIdx.y*6 + 3][1],2) +
////				   powf(shared[threadIdx.y*6 + 1][2] - shared[threadIdx.y*6 + 4][1],2) +
////				   powf(shared[threadIdx.y*6 + 2][2] - shared[threadIdx.y*6 + 5][1],2));
////		b = sqrtf(powf(shared[threadIdx.y*6 + 0][0] - shared[threadIdx.y*6 + 3][1],2) +
////				   powf(shared[threadIdx.y*6 + 1][0] - shared[threadIdx.y*6 + 4][1],2) +
////				   powf(shared[threadIdx.y*6 + 2][0] - shared[threadIdx.y*6 + 5][1],2));
//
//		a = sqrtf((shared[threadIdx.y*6 + 0][2] - shared[threadIdx.y*6 + 3][1]) * (shared[threadIdx.y*6 + 0][2] - shared[threadIdx.y*6 + 3][1]) +
//				   (shared[threadIdx.y*6 + 1][2] - shared[threadIdx.y*6 + 4][1]) * (shared[threadIdx.y*6 + 1][2] - shared[threadIdx.y*6 + 4][1]) +
//				   (shared[threadIdx.y*6 + 2][2] - shared[threadIdx.y*6 + 5][1]) * (shared[threadIdx.y*6 + 2][2] - shared[threadIdx.y*6 + 5][1]) );
//		b = sqrtf((shared[threadIdx.y*6 + 0][0] - shared[threadIdx.y*6 + 3][1]) * (shared[threadIdx.y*6 + 0][0] - shared[threadIdx.y*6 + 3][1]) +
//				   (shared[threadIdx.y*6 + 1][0] - shared[threadIdx.y*6 + 4][1]) * (shared[threadIdx.y*6 + 1][0] - shared[threadIdx.y*6 + 4][1]) +
//				   (shared[threadIdx.y*6 + 2][0] - shared[threadIdx.y*6 + 5][1]) * (shared[threadIdx.y*6 + 2][0] - shared[threadIdx.y*6 + 5][1]) );
//
//		if (a==0 && b==0){
//			P1 = 0.5;
//			P2 = 0.5;
//		}
//		else if (a!=0 && b==0){
//			P1 = 1;
//			P2 = 0;
//		}
//		else if(a==0 && b!=0){
//			P1 = 0;
//			P2 = 1;
//		}
//		else{
//			P1 = b/(a+b);
//			P2 = a/(a+b);
//		}
//
//		shared[threadIdx.y*6 + 0][1] = P1 * shared[threadIdx.y*6 + 0][2] + P2 * shared[threadIdx.y*6 + 0][0];
//		shared[threadIdx.y*6 + 1][1] = P1 * shared[threadIdx.y*6 + 1][2] + P2 * shared[threadIdx.y*6 + 1][0];
//		shared[threadIdx.y*6 + 2][1] = P1 * shared[threadIdx.y*6 + 2][2] + P2 * shared[threadIdx.y*6 + 2][0];
//	}
//	else{
//		shared[threadIdx.y*6 + 0][1] = shared[threadIdx.y*6 + 0][0];
//		shared[threadIdx.y*6 + 1][1] = shared[threadIdx.y*6 + 1][0];
//		shared[threadIdx.y*6 + 2][1] = shared[threadIdx.y*6 + 2][0];
//	}
//	__syncthreads();
//
//	//////////////////////////////////////////////////
//	/////// III. Load shared memory part ///////
//	//////////////////////////////////////////////////
//
//	if(y < width){
//		output[3*(2*width*2*x+2*(blockIdx.y*blockDim.y))+(threadIdx.y)] = shared[threadIdx.y][0];
//		output[3*(2*width*2*x+2*(blockIdx.y*blockDim.y))+(threadIdx.y+128)] = shared[threadIdx.y+128][0];
//		output[3*(2*width*2*x+2*(blockIdx.y*blockDim.y))+(threadIdx.y+256)] = shared[threadIdx.y+256][0];
//
//		output[3*(2*width*2*x+2*(blockIdx.y*blockDim.y))+(threadIdx.y+384)] = shared[threadIdx.y+384][0];
//		output[3*(2*width*2*x+2*(blockIdx.y*blockDim.y))+(threadIdx.y+512)] = shared[threadIdx.y+512][0];
//		output[3*(2*width*2*x+2*(blockIdx.y*blockDim.y))+(threadIdx.y+640)] = shared[threadIdx.y+640][0];
//
//		output[3*(2*width*(2*x+1)+2*(blockIdx.y*blockDim.y))+(threadIdx.y)] = shared[threadIdx.y][1];
//		output[3*(2*width*(2*x+1)+2*(blockIdx.y*blockDim.y))+(threadIdx.y+128)] = shared[threadIdx.y+128][1];
//		output[3*(2*width*(2*x+1)+2*(blockIdx.y*blockDim.y))+(threadIdx.y+256)] = shared[threadIdx.y+256][1];
//
//		output[3*(2*width*(2*x+1)+2*(blockIdx.y*blockDim.y))+(threadIdx.y+384)] = shared[threadIdx.y+384][1];
//		output[3*(2*width*(2*x+1)+2*(blockIdx.y*blockDim.y))+(threadIdx.y+512)] = shared[threadIdx.y+512][1];
//		output[3*(2*width*(2*x+1)+2*(blockIdx.y*blockDim.y))+(threadIdx.y+640)] = shared[threadIdx.y+640][1];
//	}
//}
//
//__global__ void ucharCopy(float* d_result_img, int height, int width, unsigned char* d_result_img_uchar){
//	int x = threadIdx.x + blockIdx.x * blockDim.x;
//	int y = threadIdx.y + blockIdx.y * blockDim.y;
//
//	if (x<=(2*height-1) && y<=(2*width-1)){
//		d_result_img_uchar[3*(x*(2*width)+y)  ] = d_result_img[3*(x*(2*width)+y)  ]; // (unsigned char)
//		d_result_img_uchar[3*(x*(2*width)+y)+1] = d_result_img[3*(x*(2*width)+y)+1];
//		d_result_img_uchar[3*(x*(2*width)+y)+2] = d_result_img[3*(x*(2*width)+y)+2];
//	}
//}
